#include <iostream>
#include <vector>
#include <cmath>
#include <stdio.h>
#include <chrono>
#include <assert.h>

#include <hip/hip_runtime.h>

using namespace std;


inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}



/* -----------------------------------------------------
                 Kernel Definitions
----------------------------------------------------- */ 

__global__
void initVector(double *X, long N)
    {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    for(int i = index; i < N; i += stride)
      {
      X[i] = cos(2.*M_PI*i/N);
      }
    }



int main(int argc,char**argv)
{
  /* -----------------------------------------------------
                  Parsing des arguments
  ----------------------------------------------------- */ 

  for(int i=0;i<argc;i++)
    cout << argv[i] << " " ;
  cout << endl;

  bool lvlOut = false;
  int  A = 1;
  int  M = 1;
  long N = 100000; 

  for(int i=0;i<argc;i++) // Parsing des options d'execution
  {
    if(argv[i]==string("-n"))
      N = stol(argv[i+1]);
    if(argv[i]==string("-m"))
      M = stol(argv[i+1]);
    if(argv[i]==string("-a"))
      A = stol(argv[i+1]);
    if(argv[i]==string("-out"))
      lvlOut = true;
  } 

  printf("Vector initialzation in parallel using CUDA:\n");



  /* -----------------------------------------------------
            Initialisation paramètres CUDA
  ----------------------------------------------------- */ 

  int deviceId;
  int numberOfSMs;

  checkCuda(hipGetDevice(&deviceId));
  checkCuda(hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));

  size_t threadsPerBlock = 128;
  size_t numberOfBlocks = 32 * numberOfSMs;

  // Allocation et initialisation des vecteurs sur le GPU
  double *X_gpu;
  
  hipMalloc(&X_gpu, sizeof(double)*N);
   
  auto begin = std::chrono::high_resolution_clock::now(); 
  
  for(int k=0;k<A;k++)
    {
    for(int i=0;i<M;i++) // Lunching 'M' kernels
      {
      initVector<<<numberOfBlocks, threadsPerBlock>>>(X_gpu, N);
      hipDeviceSynchronize();
      }
    }
  
  auto end = std::chrono::high_resolution_clock::now();
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
  
  cout << "   Vector initialization in " << elapsed.count()/(1000.0*A) << " us" << endl;

  if(lvlOut)
    {
    double *X_host;

    hipHostMalloc(&X_host, sizeof(double)*N, hipHostMallocDefault);

    hipMemcpy(X_host, X_gpu, sizeof(double)*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cout << "   Vecteur X: ";
    for(int i=0;i<N;i++)
      {
      cout << X_host[i] << " ";
      }
    cout << endl;
    hipFree(X_host);
    }

  cout << "   Completed for: " << N << " size, " << M << " repetition and averaged " << A << " times" << endl;
  
  hipFree(X_gpu);
  
  return 0 ;
}
