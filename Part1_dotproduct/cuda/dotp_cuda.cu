#include <iostream>
#include <vector>
#include <cmath>
#include <stdio.h>
#include <chrono>
#include <assert.h>

#include <hip/hip_runtime.h>
#include "hipblas.h" 

using namespace std ;

#define D_vectAdd     0
#define D_dotProduct  1

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}



/* -----------------------------------------------------
                 Kernel Definitions
----------------------------------------------------- */ 

__global__
void initVector(double *X, double *Y, long N)
    {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    for(int i = index; i < N; i += stride)
      {
      X[i] = cos(2.*M_PI*i/N);
      Y[i] = sin(2.*M_PI*i/N);
      }
    }

__global__
void elementwiseMultVect(double *X, double *Y, long N)
  {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;  

  for(int i = index; i < N; i += stride)
    {
    X[i] = X[i] * Y[i];
    }
  }

__global__
void sum_reduction_kernel_ext(double *X, long N)
  {
  uint index       = threadIdx.x + blockIdx.x * blockDim.x;
  uint stride      = blockDim.x * gridDim.x;  
  long sum_stride  = ceil(N/2.);

  for(int i = index; i < N; i += stride)
    {
      if(i+sum_stride<N)
        X[i] = X[i] + X[i+sum_stride];
    }
  }


int main(int argc,char**argv)
{
  /* -----------------------------------------------------
                  Parsing des arguments
  ----------------------------------------------------- */ 

  for(int i=0;i<argc;i++)
    cout << argv[i] << " " ;
  cout << endl;

  bool lvlOut = false;
  int  A = 1;
  int  M = 1;
  long N  = 100000; 

  for(int i=0;i<argc;i++) // Parsing des options d'execution
  {
    if(argv[i]==string("-n"))
      N = stol(argv[i+1]);
    if(argv[i]==string("-m"))
      M = stol(argv[i+1]);
    if(argv[i]==string("-a"))
      A = stol(argv[i+1]);
    if(argv[i]==string("-out"))
      lvlOut = true;
  } 

  printf("Dot product in parallel using CUDA:\n");



  /* -----------------------------------------------------
            Initialisation paramètres CUDA
  ----------------------------------------------------- */ 

  int deviceId;
  int numberOfSMs;

  checkCuda(hipGetDevice(&deviceId));
  checkCuda(hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));

  size_t threadsPerBlock = 128;
  size_t numberOfBlocks = 32 * numberOfSMs;

  hipblasHandle_t h_cuBLAS;
  hipblasCreate(&h_cuBLAS);
  hipblasSetPointerMode(h_cuBLAS, HIPBLAS_POINTER_MODE_DEVICE);

  // Allocation et initialisation des vecteurs sur le GPU
  double *X_gpu, *Y_gpu, *dotp_gpu, *dotp_host;
  
  hipMalloc(&X_gpu, sizeof(double)*N);
  hipMalloc(&Y_gpu, sizeof(double)*N);
  hipMalloc(&dotp_gpu, sizeof(double)); // Stockage du résultat de cuBLAS dot product sur le GPU
  hipHostMalloc(&dotp_host, sizeof(double), hipHostMallocDefault); // Resultat du dotp, host side
   

  /* Initialisation des vecteurs sur le gpu */

  initVector<<<numberOfBlocks, threadsPerBlock>>>(X_gpu, Y_gpu, N);
  hipDeviceSynchronize();



  /* -----------------------------------------------------
                Produit scalaire
  ----------------------------------------------------- */ 
  
  long length_reduction = N;

  auto begin = std::chrono::high_resolution_clock::now(); 
  
  for(int k=0;k<A;k++)
    {
    for(int i=0; i<M; i++) // Lunching 'repeat_trial' kernels
      {
      elementwiseMultVect<<<numberOfBlocks, threadsPerBlock>>>(X_gpu, Y_gpu, N);
      hipDeviceSynchronize();

      while(length_reduction > 1)
        {
        sum_reduction_kernel_ext<<<numberOfBlocks, threadsPerBlock>>>(X_gpu, length_reduction);
        hipDeviceSynchronize();

        length_reduction = ceil(length_reduction/2.);
        }
    
      length_reduction  = N;
      }
    }

  auto end = std::chrono::high_resolution_clock::now();
  auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);

  cout << "   Handmade dot product " << elapsed.count()/(1000.0*A) << " us" << endl;
  
  if(lvlOut)
    {
    hipMemcpy(dotp_host, X_gpu, sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cout << "      Handmade dotp = " << dotp_host[0] << endl;
    }



  /* -----------------------------------------------------
                 cuBLAS - dotProduct
  ----------------------------------------------------- */ 

  initVector<<<numberOfBlocks, threadsPerBlock>>>(X_gpu, Y_gpu, N); // Ré-initialisation de X car il a été modifié par le handmade dotp 
  hipDeviceSynchronize();

  begin = std::chrono::high_resolution_clock::now(); 

  for(int k=0;k<A;k++)
    {
    for(int i=0; i<M; i++) // Lunching 'M' kernels
      {
      hipblasDdot(h_cuBLAS, N, X_gpu, 1, Y_gpu, 1, dotp_gpu);
      hipDeviceSynchronize();
      }
    }

  end = std::chrono::high_resolution_clock::now();
  elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);  

  cout << "   cuBLAS dot product in " << elapsed.count()/(1000.0*A)<< " us" << endl;

  if(lvlOut)
    {
    hipMemcpy(dotp_host, dotp_gpu, sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cout << "      cuBLAS dotp = " << dotp_host[0] << endl;
    }

  cout << "   completed for: " << N << " size, " << M << " repetitions and averaged " << A << " times" << endl;
  
  hipFree(X_gpu);
  hipFree(Y_gpu);
  hipFree(dotp_gpu);
  hipFree(dotp_host);
  
  return 0 ;
}
