#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute capability: %d %d\n", prop.major,prop.minor);
    printf("  SM count: %d\n", prop.multiProcessorCount);
    printf("  Max threads: [%d,%d,%d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("  Max blocks: [%d,%d,%d]\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}

